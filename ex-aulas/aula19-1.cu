#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <iostream>
#include <chrono>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform_reduce.h>
#include <math.h>

struct var{
    double media;
    int N;

    var(double m, int n) : media(m) , N(n) {};
    __host__ __device__ 
    double operator()(const double &vi){
        return (vi - media)*(vi - media)/ N;
    }
};

int main() {

    std::vector<double> stocks;

    while (std::cin){
        double s;
        std::cin >> s;
        stocks.push_back(s);
    }
    int N = stocks.size();

    thrust::device_vector<double> stocks_dev(stocks);

    double soma = thrust::reduce(stocks_dev.begin(),stocks_dev.end());
    double media = soma/N;

    thrust::device_vector<double> variancias(N);

    thrust::transform(stocks_dev.begin(),stocks_dev.end(),variancias.begin(),var(media,N));

    double variancia = thrust::reduce(variancias.begin(),variancias.end(),0.0,thrust::plus<double>());


    std::cout << "Media: " << media << " Var: " << variancia << " desvio padrao: " << sqrt(variancia) << "\n";

    return 0;
}